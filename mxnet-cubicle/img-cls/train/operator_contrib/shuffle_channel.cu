#include "hip/hip_runtime.h"
#include "./shuffle_channel-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>

#define SHUFFLECHANNEL_CUDA_CHECK(condition) \
    /* Code block avoids redefinition of hipError_t error */ \
    do { \
      hipError_t error = condition; \
      CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
    }while (0)
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

namespace mshadow {
namespace cuda {
template <typename DType>
__global__ void ShuffleChannelKernel(const int count, const int feature_map_size, DType *output, 
        const DType *input, int group_row, int group_column, int len) {
    CUDA_KERNEL_LOOP(index, count) {
        const int n = index / group_row / group_column / len;
        const int i = (index / group_column / len) % group_row;
        const int j = index / len % group_column;
        const int k = index - (n * feature_map_size + (i * group_column + j) * len);
        DType* p_o = output + n * feature_map_size + (j * group_row + i) * len;
        p_o[k] = input[index];
    }
}


template <typename DType>
inline void ShuffleChannelKernelForward(const Tensor<gpu, 4, DType> &out,
                                  const Tensor<gpu, 4, DType> &data,
                                  const uint32_t group) {
    const DType *bottom_data = data.dptr_;
    DType *top_data = out.dptr_;

    const int num = data.size(0);
    const int feature_map_size = data.size(1) * data.size(2) * data.size(3);
    const int sp_sz = data.size(2) * data.size(3);
    const int chs = data.size(1);

    int group_row = group;
    int group_column = int(chs / group_row);
    CHECK_EQ(chs, (group_column * group_row)) << "Wrong group size.";

    int count = num * group_column * group_row * sp_sz;
    const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
    dim3 dimGrid(gridSize);
    dim3 dimBlock(kMaxThreadsPerBlock);
    CheckLaunchParam(dimGrid, dimBlock, "ShuffleChannelForward");
    hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
    ShuffleChannelKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(count, feature_map_size, top_data, 
            bottom_data, group_row, group_column, sp_sz);
    SHUFFLECHANNEL_CUDA_CHECK(hipPeekAtLastError());
}

template <typename DType>
inline void ShuffleChannelKernelBackward(const Tensor<gpu, 4, DType> &in_grad,
                                   const Tensor<gpu, 4, DType> &out_grad,
                                   const Tensor<gpu, 4, DType> &data,
                                   const uint32_t group) {
    const DType* top_diff = out_grad.dptr_;
    DType* bottom_diff = in_grad.dptr_;

    const int num = data.size(0);
    const int feature_map_size = data.size(1) * data.size(2) * data.size(3);
    const int sp_sz = data.size(2) * data.size(3);
    const int chs = data.size(1);

    int group_row = int(chs / group);
    int group_column = group;
    int count = num * group_column * group_row * sp_sz;
    const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
    dim3 dimGrid(gridSize);
    dim3 dimBlock(kMaxThreadsPerBlock);
    CheckLaunchParam(dimGrid, dimBlock, "ShuffleChannelBackward");
    hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
    ShuffleChannelKernel<DType><<<dimGrid, dimBlock, 0, stream>>>(count, feature_map_size, 
            bottom_diff, top_diff, group_row, group_column, sp_sz);
    SHUFFLECHANNEL_CUDA_CHECK(hipPeekAtLastError());

}
} // namespace cuda


template <typename DType>
inline void ShuffleChannelForward(const Tensor<gpu, 4, DType> &out,
                                  const Tensor<gpu, 4, DType> &data,
                                  const uint32_t group) {
    cuda::ShuffleChannelKernelForward(out, data, group);
}

template <typename DType>
inline void ShuffleChannelBackward(const Tensor<gpu, 4, DType> &in_grad,
                                   const Tensor<gpu, 4, DType> &out_grad, 
                                   const Tensor<gpu, 4, DType> &data,
                                   const uint32_t group) {
    cuda::ShuffleChannelKernelBackward(in_grad, out_grad, data, group);
}

} // namespace mshadow

namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(ShuffleChannelParam param, int dtype) {
  Operator* op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ShuffleChannelOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet


